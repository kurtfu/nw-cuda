#include "hip/hip_runtime.h"
/*****************************************************************************/
/*  HEADER INCLUDES                                                          */
/*****************************************************************************/

#include "cuda.hpp"

#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <thrust/swap.h>

/*****************************************************************************/
/*  USING DECLERATIONS                                                       */
/*****************************************************************************/

using nw::cuda;

/*****************************************************************************/
/*  NAMESPACE ALIASES                                                        */
/*****************************************************************************/

namespace cg = cooperative_groups;

/*****************************************************************************/
/*  DEVICE SYMBOLS                                                           */
/*****************************************************************************/

namespace
{
    __constant__ int nw_cuda_match;
    __constant__ int nw_cuda_miss;
    __constant__ int nw_cuda_gap;

    __constant__ std::size_t nw_cuda_n_row;
    __constant__ std::size_t nw_cuda_n_col;
}

/*****************************************************************************/
/*  DEVICE KERNELS                                                           */
/*****************************************************************************/

__device__ static void nw_cuda_fill_cell(std::size_t rw,
                                         std::size_t cl,
                                         int*        curr,
                                         int*        hv,
                                         int*        diag,
                                         char const* ref,
                                         char const* src)
{
    cg::grid_group grid   = cg::this_grid();
    std::size_t    n_vect = std::min(nw_cuda_n_row - rw, cl + 1);

    std::size_t top_row = rw;

    rw += grid.thread_rank();
    cl -= grid.thread_rank();

    if (rw - top_row >= n_vect)
    {
        return;
    }

    if (rw == 0 || cl == 0)
    {
        curr[rw] = (rw + cl) * nw_cuda_gap;
    }
    else
    {
        int eps = (ref[cl - 1] == src[rw - 1]) ? nw_cuda_match : nw_cuda_miss;

        curr[rw] = std::max({diag[rw - 1] + eps,
                             hv[rw - 1] + nw_cuda_gap,
                             hv[rw] + nw_cuda_gap});
    }
}

__device__ static void nw_cuda_fill_subad(std::size_t ad,
                                          int*        curr,
                                          int*        hv,
                                          int*        diag,
                                          char const* ref,
                                          char const* src)
{
    cg::grid_group grid = cg::this_grid();

    std::size_t rw = (ad < nw_cuda_n_col) ? 0 : ad - nw_cuda_n_col + 1;
    std::size_t cl = (ad < nw_cuda_n_col) ? ad : nw_cuda_n_col - 1;

    std::size_t n_vect = std::min(nw_cuda_n_row - rw, cl + 1);

    std::size_t n_iter = (n_vect % grid.size()) ? 1 : 0;
    n_iter += n_vect / grid.size();

    for (std::size_t iter = 0; iter < n_iter; ++iter)
    {
        nw_cuda_fill_cell(rw, cl, curr, hv, diag, ref, src);

        rw += grid.size();
        cl -= grid.size();
    }
}

__global__ static void nw_cuda_fill(std::size_t ad,
                                    int*        curr,
                                    int*        hv,
                                    int*        diag,
                                    char const* ref,
                                    char const* src)
{
    std::size_t rw = (ad < nw_cuda_n_col) ? 0 : ad - nw_cuda_n_col + 1;
    std::size_t cl = (ad < nw_cuda_n_col) ? ad : nw_cuda_n_col - 1;

    std::size_t n_vect = std::min(nw_cuda_n_row - rw, cl + 1);

    std::size_t top_row = rw;

    rw += (blockIdx.x * blockDim.x + threadIdx.x);
    cl -= (blockIdx.x * blockDim.x + threadIdx.x);

    if (rw - top_row >= n_vect)
    {
        return;
    }

    if (rw == 0 || cl == 0)
    {
        curr[rw] = (rw + cl) * nw_cuda_gap;
    }
    else
    {
        int eps = (ref[cl - 1] == src[rw - 1]) ? nw_cuda_match : nw_cuda_miss;

        curr[rw] = std::max({diag[rw - 1] + eps,
                             hv[rw - 1] + nw_cuda_gap,
                             hv[rw] + nw_cuda_gap});
    }
}

__global__ static void nw_cuda_score(int*        curr,
                                     int*        hv,
                                     int*        diag,
                                     char const* ref,
                                     char const* src)
{
    cg::grid_group grid   = cg::this_grid();
    std::size_t    n_diag = nw_cuda_n_row + nw_cuda_n_col - 1;

    for (std::size_t ad = 0; ad < n_diag; ++ad)
    {
        cg::sync(grid);

        thrust::swap(diag, hv);
        thrust::swap(hv, curr);

        nw_cuda_fill_subad(ad, curr, hv, diag, ref, src);
    }
}

/*****************************************************************************/
/*  PUBLIC METHODS                                                           */
/*****************************************************************************/

cuda::cuda(int match, int miss, int gap)
{
    int dev;
    hipGetDevice(&dev);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);

    warp_size            = prop.warpSize;
    multiprocessor_count = prop.multiProcessorCount;

    max_thread_per_block          = prop.maxThreadsPerBlock;
    max_thread_per_multiprocessor = prop.maxThreadsPerMultiProcessor;

    this->match = match;
    this->miss  = miss;
    this->gap   = gap;

    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_match), &match, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_miss), &miss, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_gap), &gap, sizeof(int));
}

int& cuda::operator()(std::vector<int>::size_type rw, std::vector<int>::size_type cl)
{
    std::size_t upper_line = std::min(n_row, n_col);
    std::size_t lower_line = std::max(n_row, n_col) - 1;

    std::size_t ad = rw + cl;

    std::size_t pos;
    std::size_t offset;

    if (ad < upper_line)
    {
        pos    = ad * (ad + 1) / 2;
        offset = rw;
    }
    else if (ad < lower_line)
    {
        std::size_t n_vect = std::min(n_row, n_col);

        pos = upper_line * (upper_line + 1) / 2;
        pos += (ad - upper_line) * n_vect;

        offset = (n_row < n_col) ? rw : n_col - cl - 1;
    }
    else
    {
        std::size_t n_diag = n_row + n_col - 1;

        ad = n_diag - ad;

        pos = (n_row * n_col) - 1;
        pos -= (ad * (ad + 1) / 2);

        offset = (n_row < n_col) ? rw : n_col - cl;
    }

    return matrix[pos + offset];
}

std::size_t cuda::row_count() const
{
    return n_row;
}

std::size_t cuda::col_count() const
{
    return n_col;
}

void cuda::fill(std::string const& ref, std::string const& src)
{
    std::size_t n_row = src.size() + 1;
    std::size_t n_col = ref.size() + 1;

    if (n_row * n_col > this->n_row * this->n_col)
    {
        matrix.reserve(n_row * n_col);
    }
    else
    {
        matrix.resize(n_row * n_col);
        matrix.shrink_to_fit();
    }

    this->n_row = n_row;
    this->n_col = n_col;

    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_n_row), &n_row, sizeof(std::size_t));
    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_n_col), &n_col, sizeof(std::size_t));

    std::size_t n_vect = std::min(n_row, n_col);

    int* d_curr;
    int* d_hv;
    int* d_diag;

    hipHostMalloc(&d_curr, n_vect * sizeof(int));
    hipHostMalloc(&d_hv, n_vect * sizeof(int));
    hipHostMalloc(&d_diag, n_vect * sizeof(int));

    char* d_ref;
    char* d_src;

    hipMalloc(&d_ref, ref.size());
    hipMemcpy(d_ref, ref.c_str(), ref.size(), hipMemcpyHostToDevice);

    hipMalloc(&d_src, src.size());
    hipMemcpy(d_src, src.c_str(), src.size(), hipMemcpyHostToDevice);

    std::size_t n_block = (n_vect % max_thread_per_block) ? 1 : 0;
    n_block += n_vect / max_thread_per_block;

    std::size_t n_thread = (n_vect % n_block) ? 1 : 0;
    n_thread += n_vect / n_block;

    if (n_thread % warp_size)
    {
        n_thread = ((n_thread / warp_size) + 1) * warp_size;
    }

    std::size_t n_diag = n_row + n_col - 1;

    for (std::size_t ad = 0; ad < n_diag; ++ad)
    {
        nw_cuda_fill<<<n_block, n_thread>>>(ad, d_curr, d_hv, d_diag, d_ref, d_src);
        hipDeviceSynchronize();

        copy_diag(ad, d_curr);

        std::swap(d_diag, d_hv);
        std::swap(d_hv, d_curr);
    }

    hipFree(d_src);
    hipFree(d_ref);

    hipHostFree(d_diag);
    hipHostFree(d_hv);
    hipHostFree(d_curr);
}

int cuda::score(std::string const& ref, std::string const& src)
{
    std::size_t n_row = src.size() + 1;
    std::size_t n_col = ref.size() + 1;

    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_n_row), &n_row, sizeof(std::size_t));
    hipMemcpyToSymbol(HIP_SYMBOL(nw_cuda_n_col), &n_col, sizeof(std::size_t));

    std::size_t n_vect = std::min(n_row, n_col);

    int* d_curr;
    int* d_hv;
    int* d_diag;

    hipMalloc(&d_curr, n_vect * sizeof(int));
    hipMalloc(&d_hv, n_vect * sizeof(int));
    hipMalloc(&d_diag, n_vect * sizeof(int));

    char* d_ref;
    char* d_src;

    hipMalloc(&d_ref, ref.size());
    hipMemcpy(d_ref, ref.c_str(), ref.size(), hipMemcpyHostToDevice);

    hipMalloc(&d_src, src.size());
    hipMemcpy(d_src, src.c_str(), src.size(), hipMemcpyHostToDevice);

    auto dimension = align_dimension(n_vect);

    std::size_t n_block  = dimension.first;
    std::size_t n_thread = dimension.second;

    void* args[] = {&d_curr, &d_hv, &d_diag, &d_ref, &d_src};

    hipLaunchCooperativeKernel((void*)nw_cuda_score, n_block, n_thread, args);
    hipDeviceSynchronize();

    int score;
    hipMemcpy(&score, &d_curr[n_row - 1], sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_ref);

    hipFree(d_diag);
    hipFree(d_hv);
    hipFree(d_curr);

    return score;
}

/*****************************************************************************/
/*  PRIVATE METHODS                                                          */
/*****************************************************************************/

std::pair<std::size_t, std::size_t> cuda::align_dimension(std::size_t n_vect)
{
    std::size_t n_block = (n_vect % max_thread_per_block) ? 1 : 0;
    n_block += n_vect / max_thread_per_block;

    if (n_block > multiprocessor_count)
    {
        n_block = multiprocessor_count;
    }

    std::size_t n_thread = (n_vect % n_block) ? 1 : 0;
    n_thread += n_vect / n_block;

    if (n_thread % warp_size)
    {
        n_thread = ((n_thread / warp_size) + 1) * warp_size;
    }

    if (n_thread > max_thread_per_multiprocessor)
    {
        n_thread = max_thread_per_multiprocessor;
    }

    return std::make_pair(n_block, n_thread);
}

void cuda::copy_diag(std::size_t ad, int* diag)
{
    std::size_t rw = (ad < n_col) ? 0 : ad - n_col + 1;
    std::size_t cl = (ad < n_col) ? ad : n_col - 1;

    std::size_t n_diag = std::min(n_row - rw, cl + 1);
    std::size_t pos    = (n_row <= n_col) ? rw : n_diag - cl - 1;

    hipMemcpy(&(*this)(rw, cl), &diag[pos], n_diag * sizeof(int), hipMemcpyDeviceToHost);
}
